#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: frenet_optimal_trajectory.cpp
	> Author: TAI Lei
	> Mail: ltai@ust.hk
	> Created Time: Wed Apr  3 09:52:17 2019
 ************************************************************************/

#include<iostream>
#include<limits>
#include<vector>
#include<opencv2/opencv.hpp>
#include<opencv2/core/core.hpp>
#include<opencv2/highgui/highgui.hpp>
#include<sys/time.h>
#include"cubic_spline.h"
#include"frenet_path.h"
#include"quintic_polynomial.h"
#include"quartic_polynomial.h"

#define SIM_LOOP 500
#define MAX_SPEED  50.0 / 3.6  // maximum speed [m/s]
#define MAX_ACCEL  2.0  // maximum acceleration [m/ss]
#define MAX_CURVATURE  1.0  // maximum curvature [1/m]
#define MAX_ROAD_WIDTH  7.0  // maximum road width [m]
#define D_ROAD_W  1.0  // road width sampling length [m]
#define DT  0.2  // time tick [s]
#define MAXT  5.0  // max prediction time [m]
#define MINT  4.0  // min prediction time [m]
#define TARGET_SPEED  30.0 / 3.6  // target speed [m/s]
#define D_T_S  5.0 / 3.6  // target speed sampling length [m/s]
#define N_S_SAMPLE  1  // sampling number of target speed
#define ROBOT_RADIUS  1.5  // robot radius [m]

#define KJ  0.1
#define KT  0.1
#define KD  1.0
#define KLAT  1.0
#define KLON  1.0

using namespace cpprobotics;


float sum_of_power(std::vector<float> value_list){
  float sum = 0;
  for(float item:value_list){
    sum += item*item;
  }
  return sum;
};

Vec_Path calc_frenet_paths(
    float c_speed, float c_d, float c_d_d, float c_d_dd, float s0){
  std::vector<FrenetPath> fp_list;
  for(float di=-1*MAX_ROAD_WIDTH; di<MAX_ROAD_WIDTH; di+=D_ROAD_W){
    for(float Ti=MINT; Ti<MAXT; Ti+=DT){
      FrenetPath fp;
      QuinticPolynomial lat_qp(c_d, c_d_d, c_d_dd, di, 0.0, 0.0, Ti);
      for(float t=0; t<Ti; t+=DT){
        fp.t.push_back(t);
        fp.d.push_back(lat_qp.calc_point(t));
        fp.d_d.push_back(lat_qp.calc_first_derivative(t));
        fp.d_dd.push_back(lat_qp.calc_second_derivative(t));
        fp.d_ddd.push_back(lat_qp.calc_third_derivative(t));
      }
      for(float tv=TARGET_SPEED - D_T_S * N_S_SAMPLE;
          tv < TARGET_SPEED + D_T_S * N_S_SAMPLE;
          tv+=D_T_S){

        FrenetPath fp_bot = fp;
        QuarticPolynomial lon_qp(s0, c_speed, 0.0, tv, 0.0, Ti);

        fp_bot.max_speed = std::numeric_limits<float>::min();
        fp_bot.max_accel = std::numeric_limits<float>::min();
        for(float t_:fp.t){
          fp_bot.s.push_back(lon_qp.calc_point(t_));
          fp_bot.s_d.push_back(lon_qp.calc_first_derivative(t_));
          fp_bot.s_dd.push_back(lon_qp.calc_second_derivative(t_));
          fp_bot.s_ddd.push_back(lon_qp.calc_third_derivative(t_));
          if(fp_bot.s_d.back() > fp_bot.max_speed){
            fp_bot.max_speed = fp_bot.s_d.back();
          }
          if(fp_bot.s_dd.back() > fp_bot.max_accel){
            fp_bot.max_accel = fp_bot.s_dd.back();
          }
        }

        float Jp = sum_of_power(fp.d_ddd);
        float Js = sum_of_power(fp_bot.s_ddd);
        float ds = (TARGET_SPEED - fp_bot.s_d.back());

        fp_bot.cd = KJ * Jp + KT * Ti + KD * std::pow(fp_bot.d.back(), 2);
        fp_bot.cv = KJ * Js + KT * Ti + KD * ds;
        fp_bot.cf = KLAT * fp_bot.cd + KLON * fp_bot.cv;

        fp_list.push_back(fp_bot);
      }
    }
  }
  return fp_list;
};

void calc_global_paths(Vec_Path & path_list, Spline2D csp){
  for (Vec_Path::iterator path_p=path_list.begin(); path_p!=path_list.end();path_p++){
    for(unsigned int i=0; i<path_p->s.size(); i++){
      if (path_p->s[i] >= csp.s.back()){
        break;
      }
      std::array<float, 2> poi = csp.calc_postion(path_p->s[i]);
      float iyaw = csp.calc_yaw(path_p->s[i]);
      float di = path_p->d[i];
      float x = poi[0] + di * std::cos(iyaw + M_PI/2.0);
      float y = poi[1] + di * std::sin(iyaw + M_PI/2.0);
      path_p->x.push_back(x);
      path_p->y.push_back(y);
    }

    for(int i=0; i<path_p->x.size()-1; i++){
      float dx = path_p->x[i + 1] - path_p->x[i];
      float dy = path_p->y[i + 1] - path_p->y[i];
      path_p->yaw.push_back(std::atan2(dy, dx));
      path_p->ds.push_back(std::sqrt(dx * dx + dy * dy));
    }

    path_p->yaw.push_back(path_p->yaw.back());
    path_p->ds.push_back(path_p->ds.back());


    path_p->max_curvature = std::numeric_limits<float>::min();
    for(int i=0; i<path_p->x.size()-1; i++){
      path_p->c.push_back((path_p->yaw[i+1]-path_p->yaw[i])/path_p->ds[i]);
      if(path_p->c.back() > path_p->max_curvature){
        path_p->max_curvature = path_p->c.back();
      }
    }
  }
};

bool check_collision(FrenetPath path, const Vec_Poi ob){
  for(auto point:ob){
    for(unsigned int i=0; i<path.x.size(); i++){
      float dist = std::pow((path.x[i] - point[0]), 2) + std::pow((path.y[i] - point[1]), 2);
      if (dist <= ROBOT_RADIUS * ROBOT_RADIUS){
        return false;
      }
    }
  }
  return true;
};

Vec_Path check_paths(Vec_Path path_list, const Vec_Poi ob){
	Vec_Path output_fp_list;
  for(FrenetPath path:path_list){
    if (path.max_speed < MAX_SPEED && path.max_accel < MAX_ACCEL && path.max_curvature < MAX_CURVATURE && check_collision(path, ob)){
      output_fp_list.push_back(path);
    }
  }
  return output_fp_list;
};

FrenetPath frenet_optimal_planning(
    Spline2D csp, float s0, float c_speed,
    float c_d, float c_d_d, float c_d_dd, Vec_Poi ob){
  Vec_Path fp_list = calc_frenet_paths(c_speed, c_d, c_d_d, c_d_dd, s0);
  calc_global_paths(fp_list, csp);
  Vec_Path save_paths = check_paths(fp_list, ob);

  float min_cost = std::numeric_limits<float>::max();
  FrenetPath final_path;
  for(auto path:save_paths){
    if (min_cost >= path.cf){
      min_cost = path.cf;
      final_path = path;
    }
  }
  return final_path;
};

cv::Point2i cv_offset(
    float x, float y, int image_width=2000, int image_height=2000){
  cv::Point2i output;
  output.x = int(x * 100) + 300;
  output.y = image_height - int(y * 100) - image_height/3;
  return output;
};

int main(){
  Vec_f wx({0.0, 10.0, 20.5, 35.0, 70.5});
  Vec_f wy({0.0, -6.0, 5.0, 6.5, 0.0});
  std::vector<Poi_f> obstcles{
    {{20.0, 10.0}},
    {{30.0, 6.0}},
    {{30.0, 8.0}},
    {{35.0, 8.0}},
    {{50.0, 3.0}}
  };

  Spline2D csp_obj(wx, wy);
  Vec_f r_x;
  Vec_f r_y;
  Vec_f ryaw;
  Vec_f rcurvature;
  Vec_f rs;

  for(float i=0; i<csp_obj.s.back(); i+=0.1){
    std::array<float, 2> point_ = csp_obj.calc_postion(i);
    r_x.push_back(point_[0]);
    r_y.push_back(point_[1]);
    ryaw.push_back(csp_obj.calc_yaw(i));
    rcurvature.push_back(csp_obj.calc_curvature(i));
    rs.push_back(i);
  }

  float c_speed = 10.0/3.6;
  float c_d = 2.0;
  float c_d_d = 0.0;
  float c_d_dd = 0.0;
  float s0 = 0.0;

  float area = 20.0;

  cv::namedWindow("frenet", cv::WINDOW_NORMAL);
  int count = 0;

  for(int i=0; i<SIM_LOOP; i++){
    FrenetPath final_path = frenet_optimal_planning(
      csp_obj, s0, c_speed, c_d, c_d_d, c_d_dd, obstcles);
    s0 = final_path.s[1];
    c_d = final_path.d[1];
    c_d_d = final_path.d_d[1];
    c_d_dd = final_path.d_dd[1];
    c_speed = final_path.s_d[1];

    if (std::pow((final_path.x[1] - r_x.back()), 2) + std::pow((final_path.y[1]-r_y.back()), 2) <= 1.0){
        break;
    }

    // visualization
    cv::Mat bg(2000, 8000, CV_8UC3, cv::Scalar(255, 255, 255));
    for(unsigned int i=1; i<r_x.size(); i++){
      cv::line(
        bg,
        cv_offset(r_x[i-1], r_y[i-1], bg.cols, bg.rows),
        cv_offset(r_x[i], r_y[i], bg.cols, bg.rows),
        cv::Scalar(0, 0, 0),
        10);
    }
    for(unsigned int i=0; i<final_path.x.size(); i++){
      cv::circle(
        bg,
        cv_offset(final_path.x[i], final_path.y[i], bg.cols, bg.rows),
        40, cv::Scalar(255, 0, 0), -1);
    }

    cv::circle(
      bg,
      cv_offset(final_path.x.front(), final_path.y.front(), bg.cols, bg.rows),
      50, cv::Scalar(0, 255, 0), -1);

    for(unsigned int i=0; i<obstcles.size(); i++){
      cv::circle(
        bg,
        cv_offset(obstcles[i][0], obstcles[i][1], bg.cols, bg.rows),
        40, cv::Scalar(0, 0, 255), 5);
    }

    cv::putText(
      bg,
      "Speed: " + std::to_string(c_speed*3.6).substr(0, 4) + "km/h",
      cv::Point2i((int)bg.cols*0.5, (int)bg.rows*0.1),
      cv::FONT_HERSHEY_SIMPLEX,
      5,
      cv::Scalar(0, 0, 0),
      10);


    cv::imshow("frenet", bg);
    cv::waitKey(5);

    // save image in build/bin/pngs
    // struct timeval tp;
    // gettimeofday(&tp, NULL);
    // long int ms = tp.tv_sec * 1000 + tp.tv_usec / 1000;
    // std::string int_count = std::to_string(ms);
    // cv::imwrite("./pngs/"+int_count+".png", bg);
  }
  return 0;
};
