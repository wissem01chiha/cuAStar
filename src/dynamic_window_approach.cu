#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: main.cpp
	> Author: TAI Lei
	> Mail: ltai@ust.hk
	> Created Time: Thu Mar  7 19:39:14 2019
 ************************************************************************/

#include<iostream>
#include<vector>
#include<array>
#include<cmath>
#include<opencv2/opencv.hpp>
#include<opencv2/core/core.hpp>
#include<opencv2/highgui/highgui.hpp>

#define PI 3.141592653

using Traj = std::vector<std::array<float, 5>>;
using Obstacle = std::vector<std::array<float, 2>>;
using State = std::array<float, 5>;
using Window = std::array<float, 4>;
using Point = std::array<float, 2>;
using Control = std::array<float, 2>;

class Config{
public:
  float max_speed = 1.0;
  float min_speed = -0.5;
  float max_yawrate = 40.0 * PI / 180.0;
  float max_accel = 0.2;
  float robot_radius = 1.0;
  float max_dyawrate = 40.0 * PI / 180.0;

  float v_reso = 0.01;
  float yawrate_reso = 0.1 * PI / 180.0;

  float dt = 0.1;
  float predict_time = 3.0;
  float to_goal_cost_gain = 1.0;
  float speed_cost_gain = 1.0;
};

State motion(State x, Control u, float dt){
  x[2] += u[1] * dt;
  x[0] += u[0] * std::cos(x[2]) * dt;
  x[1] += u[0] * std::sin(x[2]) * dt;
  x[3] = u[0];
  x[4] = u[1];
  return x;
};

Window calc_dynamic_window(State x, Config config){

  return {{
    std::max((x[3] - config.max_accel * config.dt), config.min_speed),
    std::min((x[3] + config.max_accel * config.dt), config.max_speed),
    std::max((x[4] - config.max_dyawrate * config.dt), -config.max_yawrate),
    std::min((x[4] + config.max_dyawrate * config.dt), config.max_yawrate)
  }};
};


Traj calc_trajectory(State x, float v, float y, Config config){

  Traj traj;
  traj.push_back(x);
  float time = 0.0;
  while (time <= config.predict_time){
    x = motion(x, std::array<float, 2>{{v, y}}, config.dt);
    traj.push_back(x);
    time += config.dt;
  }
  return traj;
};


float calc_obstacle_cost(Traj traj, Obstacle ob, Config config){
  // calc obstacle cost inf: collistion, 0:free
  int skip_n = 2;
  float minr = std::numeric_limits<float>::max();

  for (unsigned int ii=0; ii<traj.size(); ii+=skip_n){
    for (unsigned int i=0; i< ob.size(); i++){
      float ox = ob[i][0];
      float oy = ob[i][1];
      float dx = traj[ii][0] - ox;
      float dy = traj[ii][1] - oy;

      float r = std::sqrt(dx*dx + dy*dy);
      if (r <= config.robot_radius){
          return std::numeric_limits<float>::max();
      }

      if (minr >= r){
          minr = r;
      }
    }
  }

  return 1.0 / minr;
};

float calc_to_goal_cost(Traj traj, Point goal, Config config){

  float goal_magnitude = std::sqrt(goal[0]*goal[0] + goal[1]*goal[1]);
  float traj_magnitude = std::sqrt(std::pow(traj.back()[0], 2) + std::pow(traj.back()[1], 2));
  float dot_product = (goal[0] * traj.back()[0]) + (goal[1] * traj.back()[1]);
  float error = dot_product / (goal_magnitude * traj_magnitude);
  float error_angle = std::acos(error);
  float cost = config.to_goal_cost_gain * error_angle;

  return cost;
};

Traj calc_final_input(
  State x, Control& u,
  Window dw, Config config, Point goal,
  std::vector<std::array<float, 2>>ob){

    float min_cost = 10000.0;
    Control min_u = u;
    min_u[0] = 0.0;
    Traj best_traj;

    // evalucate all trajectory with sampled input in dynamic window
    for (float v=dw[0]; v<=dw[1]; v+=config.v_reso){
      for (float y=dw[2]; y<=dw[3]; y+=config.yawrate_reso){

        Traj traj = calc_trajectory(x, v, y, config);

        float to_goal_cost = calc_to_goal_cost(traj, goal, config);
        float speed_cost = config.speed_cost_gain * (config.max_speed - traj.back()[3]);
        float ob_cost = calc_obstacle_cost(traj, ob, config);
        float final_cost = to_goal_cost + speed_cost + ob_cost;

        if (min_cost >= final_cost){
          min_cost = final_cost;
          min_u = Control{{v, y}};
          best_traj = traj;
        }
      }
    }
    u = min_u;
    return best_traj;
};


Traj dwa_control(State x, Control & u, Config config,
  Point goal, Obstacle ob){
    // # Dynamic Window control
    Window dw = calc_dynamic_window(x, config);
    Traj traj = calc_final_input(x, u, dw, config, goal, ob);

    return u, traj;
  }

cv::Point2i cv_offset(
    float x, float y, int image_width=2000, int image_height=2000){
  cv::Point2i output;
  output.x = int(x * 100) + image_width/2;
  output.y = image_height - int(y * 100) - image_height/3;
  return output;
};


int main(){
  State x({{0.0, 0.0, PI/8.0, 0.0, 0.0}});
  Point goal({{10.0,10.0}});
  Obstacle ob({
    {{-1, -1}},
    {{0, 2}},
    {{4.0, 2.0}},
    {{5.0, 4.0}},
    {{5.0, 5.0}},
    {{5.0, 6.0}},
    {{5.0, 9.0}},
    {{8.0, 9.0}},
    {{7.0, 9.0}},
    {{12.0, 12.0}}
  });

  Control u({{0.0, 0.0}});
  Config config;
  Traj traj;
  traj.push_back(x);

  bool terminal = false;

  cv::namedWindow("dwa", cv::WINDOW_NORMAL);
  int count = 0;

  for(int i=0; i<1000 && !terminal; i++){
    Traj ltraj = dwa_control(x, u, config, goal, ob);
    x = motion(x, u, config.dt);
    traj.push_back(x);


    // visualization
    cv::Mat bg(3500,3500, CV_8UC3, cv::Scalar(255,255,255));
    cv::circle(bg, cv_offset(goal[0], goal[1], bg.cols, bg.rows),
               30, cv::Scalar(255,0,0), 5);
    for(unsigned int j=0; j<ob.size(); j++){
      cv::circle(bg, cv_offset(ob[j][0], ob[j][1], bg.cols, bg.rows),
                 20, cv::Scalar(0,0,0), -1);
    }
    for(unsigned int j=0; j<ltraj.size(); j++){
      cv::circle(bg, cv_offset(ltraj[j][0], ltraj[j][1], bg.cols, bg.rows),
                 7, cv::Scalar(0,255,0), -1);
    }
    cv::circle(bg, cv_offset(x[0], x[1], bg.cols, bg.rows),
               30, cv::Scalar(0,0,255), 5);


    cv::arrowedLine(
      bg,
      cv_offset(x[0], x[1], bg.cols, bg.rows),
      cv_offset(x[0] + std::cos(x[2]), x[1] + std::sin(x[2]), bg.cols, bg.rows),
      cv::Scalar(255,0,255),
      7);

    if (std::sqrt(std::pow((x[0] - goal[0]), 2) + std::pow((x[1] - goal[1]), 2)) <= config.robot_radius){
      terminal = true;
      for(unsigned int j=0; j<traj.size(); j++){
        cv::circle(bg, cv_offset(traj[j][0], traj[j][1], bg.cols, bg.rows),
                    7, cv::Scalar(0,0,255), -1);
      }
    }


    cv::imshow("dwa", bg);
    cv::waitKey(5);

    // std::string int_count = std::to_string(count);
    // cv::imwrite("./pngs/"+std::string(5-int_count.length(), '0').append(int_count)+".png", bg);

    count++;
  }
}
