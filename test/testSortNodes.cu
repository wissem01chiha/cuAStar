#define CUASTAR_DEBUG
#include <hip/hip_runtime.h>
#include "../include/cuAStar.hpp"

template <typename T>
void printNodes(Node3d<T>* nodes, int N) {
    for (int i = 0; i < N; ++i) {
        std::cout << "Node " << i << ": ( " << nodes[i].x << ", " << nodes[i].y << ", " << nodes[i].z << ")\n";
    }
}

#define CUDA_CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in file '" << __FILE__ << "' in line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

int main() {
    const int N = 2024;  
    const int axis = 1;  


    Node3d<float> h_nodes[N];
    for (int i = 0; i < N; ++i) {
        h_nodes[i] = Node3d<float>(rand() / (float)RAND_MAX, rand() / (float)RAND_MAX, rand() / (float)RAND_MAX);
    }

    Node3d<float> *d_nodesArray, *d_nodesArraySorted;
    CUDA_CHECK(hipMalloc((void**)&d_nodesArray, N * sizeof(Node3d<float>)));
    CUDA_CHECK(hipMalloc((void**)&d_nodesArraySorted, N * sizeof(Node3d<float>)));

    CUDA_CHECK(hipMemcpy(d_nodesArray, h_nodes, N * sizeof(Node3d<float>), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;


    CUDA_CHECK(hipEventRecord(start, 0));

    enumerationSortNodes<Node3d<float>, float><<<blocks, threadsPerBlock>>>(d_nodesArray, N, axis, d_nodesArraySorted);


    CUDA_CHECK(hipPeekAtLastError());

    CUDA_CHECK(hipEventRecord(stop, 0));

    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));


    CUDA_CHECK(hipMemcpy(h_nodes, d_nodesArraySorted, N * sizeof(Node3d<float>), hipMemcpyDeviceToHost));

    std::cout << "Sorted Nodes:\n";
    printNodes(h_nodes, N);

    CUDA_CHECK(hipFree(d_nodesArray));
    CUDA_CHECK(hipFree(d_nodesArraySorted));

    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}
