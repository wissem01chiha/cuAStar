#include "hip/hip_runtime.h"
/**
 * @file test_math.cu 
 */
#define ENABLE_CUDA_ARCH 1
#include "../include/math.hpp"
#include <iostream>

__global__ void ker_vec_diff(const double* d_input,double* d_output,int32_t n){
    vec_diff(d_input, d_output, n);
    __syncthreads();
};

__global__ void ker_quadratic_interpolation(double* result_array,double* x, 
                                      double* y){
    quadratic_interpolation(result_array,x,y);
    __syncthreads();
};

__global__ void ker_cum_sum(const double* input, double* output, int32_t n){
    cum_sum(input, output,n);
    __syncthreads();
};

__global__ void interp_refer(const double* params, double x, double* result){
  *result =  params[0] * x * x + params[1] * x + params[2];
};


int main() {
    const int32_t n = 4;
    double h_input[n] = {1.0, 2.0, 3.0, 4.0};
    double h_output[n] = {2};

    double* d_input;
    hipMalloc(&d_input, n * sizeof(double));

    double* d_output;
    hipMalloc(&d_output, n * sizeof(double));

    hipMemcpy(d_input, h_input, n * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = 3;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    ker_vec_diff<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, n);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, n * sizeof(double), hipMemcpyDeviceToHost);

    std::cout << "Input values: ";
    for (int i = 0; i < n; i++) {
        std::cout << h_input[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Output values: ";
    for (int i = 0; i < n; i++) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    double expected_output[n] = {0.0, 1.0, 1.0, 1.0};
    for (int i = 1; i < n; i++) {
        if (h_output[i] == expected_output[i]) {
            std::cout << "Value at index " << i << " is correct: " << h_output[i] << std::endl;
        } else {
            std::cout << "Value at index " << i << " is incorrect: " << h_output[i] << std::endl;
        }
    }
    hipFree(d_input);
    hipFree(d_output);
    return 0;
};
